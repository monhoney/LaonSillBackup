#include "hip/hip_runtime.h"
/*
 * PoolingLayer.cpp
 *
 *  Created on: 2016. 5. 23.
 *      Author: jhkim
 */

#include "PoolingLayer.h"
#include "PropMgmt.h"
#include "MemoryMgmt.h"
#include "StdOutLog.h"

#define POOLINGLAYER_LOG 0

using namespace std;

template <typename Dtype>
void PoolingLayer<Dtype>::reshape() {
	Layer<Dtype>::_adjustInputShape();

	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	uint32_t batches 	= inputShape[0];
	uint32_t channels 	= inputShape[1];
	uint32_t rows 		= inputShape[2];
	uint32_t cols 		= inputShape[3];


	if (this->globalPooling) {
		this->poolDim.rows = rows;
		this->poolDim.cols = cols;
	}

	if (this->pooling_fn) {
		PoolingFactory<Dtype>::destroy(this->pooling_fn);
	}
	this->pooling_fn = PoolingFactory<Dtype>::create(this->poolingType, this->poolDim);


	checkCUDNN(hipdnnSetTensor4dDescriptor(
			this->inputTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batches, channels, rows, cols));

	//int n = 0, c = 0, h = 0, w = 0;
	/*
	checkCUDNN(hipdnnGetPooling2dForwardOutputDim(
			this->pooling_fn->getPoolDesc(),
			this->inputTensorDesc,
			&n, &c, &h, &w));
			*/


	int pooledHeight = static_cast<int>(ceil(static_cast<float>(
			rows + 2 * this->poolDim.pad - this->poolDim.rows) / this->poolDim.stride)) + 1;
	int pooledWidth = static_cast<int>(ceil(static_cast<float>(
			cols + 2 * this->poolDim.pad - this->poolDim.cols) / this->poolDim.stride)) + 1;

	if (this->poolDim.pad) {
		if ((pooledHeight - 1) * this->poolDim.stride >= rows + this->poolDim.pad) {
			pooledHeight--;
		}
		if ((pooledWidth - 1) * this->poolDim.stride >= cols + this->poolDim.pad) {
			pooledWidth--;
		}
		assert((pooledHeight - 1) * this->poolDim.stride < rows + this->poolDim.pad);
		assert((pooledWidth - 1) * this->poolDim.stride < cols + this->poolDim.pad);
	}

	checkCUDNN(hipdnnSetTensor4dDescriptor(
			this->outputTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batches, channels, pooledHeight, pooledWidth));




#if POOLINGLAYER_LOG
	uint32_t obatches = static_cast<uint32_t>(batches);
	uint32_t ochannels = static_cast<uint32_t>(channels);
	uint32_t orows = static_cast<uint32_t>(pooledHeight);
	uint32_t ocols = static_cast<uint32_t>(pooledWidth);

	printf("<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n",
			this->getName().c_str(), obatches, ochannels, orows, ocols);
#endif

	this->_inputShape[0] = inputShape;
	this->_outputData[0]->reshape({
		static_cast<uint32_t>(batches),
		static_cast<uint32_t>(channels),
		static_cast<uint32_t>(pooledHeight),
		static_cast<uint32_t>(pooledWidth)});

	/*
	this->setInDimension(this->_inputData[0]->getShape());

	hipdnnTensorDescriptor_t tempInputTensorDesc;
	checkCUDNN(hipdnnCreateTensorDescriptor(&tempInputTensorDesc));
	checkCUDNN(hipdnnSetTensor4dDescriptor(tempInputTensorDesc,
				HIPDNN_TENSOR_NCHW,
				HIPDNN_DATA_FLOAT,
				this->in_dim.batches, this->in_dim.channels, this->in_dim.rows,
                this->in_dim.cols));

	int n, c, h, w;
	checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pooling_fn->getPoolDesc(),
			tempInputTensorDesc,
			&n, &c, &h, &w));

	this->out_dim.batches = n;
	this->out_dim.channels = c;
	this->out_dim.rows = h;
	this->out_dim.cols = w;

	checkCUDNN(hipdnnDestroyTensorDescriptor(tempInputTensorDesc));

	if(recursive) {
		Layer<Dtype>::_shape();
	}
	*/
}

template <typename Dtype>
void PoolingLayer<Dtype>::feedforward() {
	reshape();

	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

	this->pooling_fn->forward(this->inputTensorDesc, d_inputData,
			this->outputTensorDesc, d_outputData);
}

template <typename Dtype>
void PoolingLayer<Dtype>::backpropagation() {
	if (SLPROP_BASE(propDown)[0]) {
		const Dtype* d_outputData = this->_outputData[0]->device_data();
		const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
		const Dtype* d_inputData = this->_inputData[0]->device_data();
		Dtype* d_inputGrad = this->_inputData[0]->mutable_device_grad();
		this->pooling_fn->backward(this->outputTensorDesc, d_outputData, d_outputGrad,
				this->inputTensorDesc, d_inputData, d_inputGrad);
	}
}

template void PoolingLayer<float>::reshape();
template void PoolingLayer<float>::feedforward();
template void PoolingLayer<float>::backpropagation();
